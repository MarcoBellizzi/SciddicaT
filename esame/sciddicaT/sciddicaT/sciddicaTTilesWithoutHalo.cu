#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "util.hpp"
#include <hip/hip_runtime.h>

using namespace std;

// ----------------------------------------------------------------------------
// I/O parameters used to index argv[]
// ----------------------------------------------------------------------------
#define HEADER_PATH_ID 1
#define DEM_PATH_ID 2
#define SOURCE_PATH_ID 3
#define OUTPUT_PATH_ID 4
#define STEPS_ID 5
// ----------------------------------------------------------------------------
// Simulation parameters
// ----------------------------------------------------------------------------
#define P_R 0.5
#define P_EPSILON 0.001
#define ADJACENT_CELLS 4
#define STRLEN 256

#define TILE_SIZE_O 10

// ----------------------------------------------------------------------------
// Read/Write access macros linearizing single/multy layer buffer 2D indices
// ----------------------------------------------------------------------------
#define SET(M, columns, i, j, value) ((M)[(((i) * (columns)) + (j))] = (value))
#define GET(M, columns, i, j) (M[(((i) * (columns)) + (j))])
#define BUF_SET(M, rows, columns, n, i, j, value) ( (M)[( ((n)*(rows)*(columns)) + ((i)*(columns)) + (j) )] = (value) )
#define BUF_GET(M, rows, columns, n, i, j) ( M[( ((n)*(rows)*(columns)) + ((i)*(columns)) + (j) )] )

// ----------------------------------------------------------------------------
// I/O functions
// ----------------------------------------------------------------------------
void readHeaderInfo(char* path, int &nrows, int &ncols, /*double &xllcorner, double &yllcorner, double &cellsize,*/ double &nodata)
{
  FILE* f;
  
  if ( (f = fopen(path,"r") ) == 0){
    printf("%s configuration header file not found\n", path);
    exit(0);
  }

  //Reading the header
  char str[STRLEN];
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); ncols = atoi(str);      //ncols
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); nrows = atoi(str);      //nrows
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); //xllcorner = atof(str);  //xllcorner
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); //yllcorner = atof(str);  //yllcorner
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); //cellsize = atof(str);   //cellsize
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); nodata = atof(str);     //NODATA_value 
}

bool loadGrid2D(double *M, int rows, int columns, char *path)
{
  FILE *f = fopen(path, "r");

  if (!f) {
    printf("%s grid file not found\n", path);
    exit(0);
  }

  char str[STRLEN];
  for (int i = 0; i < rows; i++)
    for (int j = 0; j < columns; j++)
    {
      fscanf(f, "%s", str);
      SET(M, columns, i, j, atof(str));
    }

  fclose(f);

  return true;
}

bool saveGrid2Dr(double *M, int rows, int columns, char *path)
{
  FILE *f;
  f = fopen(path, "w");

  if (!f)
    return false;

  char str[STRLEN];
  for (int i = 0; i < rows; i++)
  {
    for (int j = 0; j < columns; j++)
    {
      sprintf(str, "%f ", GET(M, columns, i, j));
      fprintf(f, "%s ", str);
    }
    fprintf(f, "\n");
  }

  fclose(f);

  return true;
}

// ----------------------------------------------------------------------------
// init kernel, called once before the simulation loop
// ----------------------------------------------------------------------------
__global__ void sciddicaTSimulationInit_Kernel(int r, int c, double* Sz, double* Sh, int i_start, int i_end, int j_start, int j_end)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;    //righe
  int j = blockIdx.y * blockDim.y + threadIdx.y;    //colonne
  if(i < i_start || i >= i_end)
    return;
  if(j < j_start || j >= j_end)
    return;
 
  double z, h;
  h = GET(Sh, c, i, j);

  if (h > 0.0)
  {
    z = GET(Sz, c, i, j);
    SET(Sz, c, i, j, z - h);
  }
}

// ----------------------------------------------------------------------------
// computing kernels, aka elementary processes in the XCA terminology
// ----------------------------------------------------------------------------
__global__ void sciddicaTResetFlows_Kernel(int r, int c, double nodata, double* Sf, int i_start, int i_end, int j_start, int j_end)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if(i < i_start || i >= i_end)
    return;
  if(j < j_start || j >= j_end)
    return;

  BUF_SET(Sf, r, c, 0, i, j, 0.0);
  BUF_SET(Sf, r, c, 1, i, j, 0.0);
  BUF_SET(Sf, r, c, 2, i, j, 0.0);
  BUF_SET(Sf, r, c, 3, i, j, 0.0);
}

__global__ void sciddicaTFlowsComputation_Kernel(int r, int c, double nodata, int* Xi, int* Xj, double *Sz, double *Sh, double *Sf, double p_r, double p_epsilon, int i_start, int i_end, int j_start, int j_end)
{
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int i = blockIdx.x * TILE_SIZE_O + tx;
  int j = blockIdx.y * TILE_SIZE_O + ty;

  __shared__ double Sz_shared[TILE_SIZE_O][TILE_SIZE_O];
  __shared__ double Sh_shared[TILE_SIZE_O][TILE_SIZE_O];

  Sz_shared[tx][ty] = GET(Sz, c, i, j);
  Sh_shared[tx][ty] = GET(Sh, c, i, j);

  __syncthreads();

  if(i < i_start || i >= i_end)
    return;
  if(j < j_start || j >= j_end)
    return;
  
  bool eliminated_cells[5] = {false, false, false, false, false};
  bool again;
  double cells_count;
  double average;
  double m;
  double u[5];
  int n;
  double z, h;

  m = Sh_shared[tx][ty] - p_epsilon;
  u[0] = Sz_shared[tx][ty] + p_epsilon;

  for(int k = 1; k < 5; k++)
  {
    if(tx + Xi[k] < 0 || tx + Xi[k] >= TILE_SIZE_O || ty + Xj[k] < 0 || ty + Xj[k] >= TILE_SIZE_O)
    {
      z = GET(Sz, c, i + Xi[k], j + Xj[k]);
      h = GET(Sh, c, i + Xi[k], j + Xj[k]);
    }
    else
    {
      z = Sz_shared[tx + Xi[k]][ty + Xj[k]];
      h = Sh_shared[tx + Xi[k]][ty + Xj[k]];
    }  
    u[k] = z + h;
  }

  do
  {
    again = false;
    average = m;
    cells_count = 0;

    for (n = 0; n < 5; n++)
      if (!eliminated_cells[n])
      {
        average += u[n];
        cells_count++;
      }

    if (cells_count != 0)
      average /= cells_count;

    for (n = 0; n < 5; n++)
    {
      if ((average <= u[n]) && (!eliminated_cells[n]))
      {
        eliminated_cells[n] = true;
        again = true;
      }
    }
  } while (again);

  if (!eliminated_cells[1]) BUF_SET(Sf, r, c, 0, i, j, (average - u[1]) * p_r);
  if (!eliminated_cells[2]) BUF_SET(Sf, r, c, 1, i, j, (average - u[2]) * p_r);
  if (!eliminated_cells[3]) BUF_SET(Sf, r, c, 2, i, j, (average - u[3]) * p_r);
  if (!eliminated_cells[4]) BUF_SET(Sf, r, c, 3, i, j, (average - u[4]) * p_r);
}

__global__ void sciddicaTWidthUpdate_Kernel(int r, int c, double nodata, int* Xi, int* Xj, double *Sz, double *Sh, double *Sf, int i_start, int i_end, int j_start, int j_end)
{
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  __shared__ double shared[TILE_SIZE_O][TILE_SIZE_O][4];
 
  shared[tx][ty][0] = BUF_GET(Sf, r, c, 0, i, j);
  shared[tx][ty][1] = BUF_GET(Sf, r, c, 1, i, j);
  shared[tx][ty][2] = BUF_GET(Sf, r, c, 2, i, j);
  shared[tx][ty][3] = BUF_GET(Sf, r, c, 3, i, j);

  __syncthreads();

  if(i < i_start || i >= i_end)
    return;
  if(j < j_start || j >= j_end)
    return;
  
  double h_next;
  h_next = GET(Sh, c, i, j);

  for(int k = 1, k_inv = 3; k_inv >= 0; k++, k_inv = k_inv-1)
  {
    if(tx + Xi[k] < 0 || tx + Xi[k] >= TILE_SIZE_O || ty + Xj[k] < 0 || ty + Xj[k] >= TILE_SIZE_O)
    {
      h_next += BUF_GET(Sf, r, c, k_inv, i+Xi[k], j+Xj[k]) - shared[tx][ty][k-1];
    }
    else
    {
      h_next += shared[tx + Xi[k]][ty + Xj[k]][k_inv] - shared[tx][ty][k-1];
    }
  }

  SET(Sh, c, i, j, h_next );
}


// ----------------------------------------------------------------------------
// Function main()
// ----------------------------------------------------------------------------
int main(int argc, char **argv)
{
  int rows, cols;
  double nodata;
  readHeaderInfo(argv[HEADER_PATH_ID], rows, cols, nodata);

  int r = rows;                  // r: grid rows
  int c = cols;                  // c: grid columns
  int i_start = 1, i_end = r-1;  // [i_start,i_end[: kernels application range along the rows
  int j_start = 1, j_end = c-1;  // [i_start,i_end[: kernels application range along the rows
  double *Sz;                    // Sz: substate (grid) containing the cells' altitude a.s.l.
  double *Sh;                    // Sh: substate (grid) containing the cells' flow thickness
  double *Sf;                    // Sf: 4 substates containing the flows towards the 4 neighs
  int *Xi;                       // Xi: von Neuman neighborhood row coordinates (see below)
  int *Xj;                       // Xj: von Neuman neighborhood col coordinates (see below)
  double p_r = P_R;                 // p_r: minimization algorithm outflows dumping factor
  double p_epsilon = P_EPSILON;     // p_epsilon: frictional parameter threshold
  int steps = atoi(argv[STEPS_ID]); //steps: simulation steps

  hipMallocManaged(&Xi, sizeof(int) * 5);
  hipMallocManaged(&Xj, sizeof(int) * 5);

  Xi[0] = 0;
  Xi[1] = -1;
  Xi[2] = 0;
  Xi[3] = 0;
  Xi[4] = 1;

  Xj[0] = 0;
  Xj[1] = 0;
  Xj[2] = -1;
  Xj[3] = 1;
  Xj[4] = 0;

  hipMallocManaged(&Sz, sizeof(double) * r * c);
  hipMallocManaged(&Sh, sizeof(double) * r * c);
  hipMallocManaged(&Sf, sizeof(double) * ADJACENT_CELLS * r * c);

  loadGrid2D(Sz, r, c, argv[DEM_PATH_ID]);    // Load Sz from file
  loadGrid2D(Sh, r, c, argv[SOURCE_PATH_ID]); // Load Sh from file

  //int block_size = 512;
  //int number_of_blocks = ceil(r*c/block_size);

  dim3 dimGrid(ceil(r/(float)(TILE_SIZE_O + 1)), ceil(c/(float)(TILE_SIZE_O + 1)), 1);
  dim3 dimBlock(TILE_SIZE_O,TILE_SIZE_O,1);

  // Init kernel
  sciddicaTSimulationInit_Kernel<<<dimGrid, dimBlock>>>(r, c, Sz, Sh, i_start, i_end, j_start, j_end);
  hipDeviceSynchronize();

  util::Timer cl_timer;
  
  // Simulation loop
  for(int s = 0; s < steps; ++s)
  {
    // ResetFlow kernel
    sciddicaTResetFlows_Kernel<<<dimGrid, dimBlock>>>(r, c, nodata, Sf, i_start, i_end, j_start, j_end);
    hipDeviceSynchronize();

    // FlowComputation kernel
    sciddicaTFlowsComputation_Kernel<<<dimGrid, dimBlock>>>(r, c, nodata, Xi, Xj, Sz, Sh, Sf, p_r, p_epsilon, i_start, i_end, j_start, j_end);
    hipDeviceSynchronize();

    // WidthUpdate kernel
    sciddicaTWidthUpdate_Kernel<<<dimGrid, dimBlock>>>(r, c, nodata, Xi, Xj, Sz, Sh, Sf, i_start, i_end, j_start, j_end);
    hipDeviceSynchronize();
  }
  double cl_time = static_cast<double>(cl_timer.getTimeMilliseconds()) / 1000.0;
  printf("Elapsed time: %lf [s]\n", cl_time);

  saveGrid2Dr(Sh, r, c, argv[OUTPUT_PATH_ID]); // Save Sh to file

  printf("Releasing memory...\n");
  
  hipFree(Sz);
  hipFree(Sh);
  hipFree(Sf);
  
  hipFree(Xi);
  hipFree(Xj);

  return 0;
}